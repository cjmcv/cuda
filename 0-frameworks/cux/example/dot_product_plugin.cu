#include "hip/hip_runtime.h"
#include "operator/kernel_interface.h"
#include "executor.h"

__global__ void DotKernel(const int len, const float *vec_a, const float *vec_b, float *res) {
  // Prevents memory access across the border.
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
    i < len;
    i += blockDim.x * gridDim.x) {
    //__shared__ float smem[BLOCK_SIZE];
    extern __shared__ float smem[]; // Dynamic allocation.
    smem[threadIdx.x] = vec_a[i] * vec_b[i];
    __syncthreads();

    // Summarize in blocks.
    // Limiting conditions: len should be a multiple of block_size.
    int count = blockDim.x / 2;
    while (count >= 1) {
      if (threadIdx.x < count) {
        smem[threadIdx.x] += smem[count + threadIdx.x];
      }
      // Synchronize the threads within the block,
      // then go to next round together.
      __syncthreads();
      count /= 2;
    }

    if (threadIdx.x == 0)
      atomicAdd(res, smem[0]);
  }
}

cux::KernelInterface *DotProductGPUPlugin() {
  using namespace cux;

  auto get_config = [&](int len) -> Config1D {
    Config1D config;
    config.threads_per_block = 1024;
    config.blocks_per_grid = (len + config.threads_per_block - 1) / config.threads_per_block;
    config.shared_memory_size = config.threads_per_block * sizeof(float);
    return config;
  };

  auto func = [&](Config1D config, int len, const void *vec_a, const void *vec_b, void *res) -> void {
    DotKernel << <config.blocks_per_grid,
      config.threads_per_block,
      config.shared_memory_size >> >
      (len, (float *)vec_a, (float *)vec_b, (float *)res);
  };

  DotGpuKernelIF *kernel = new DotGpuKernelIF();
  kernel->type_flag = TypeFlag::FLOAT32;
  kernel->describe_info = "Plugin example: It's the same as kernel V0";
  kernel->get_config = get_config;
  kernel->func = func;
  kernel->config_kernel = DotKernel;

  return kernel;
}