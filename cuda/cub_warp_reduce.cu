#include "hip/hip_runtime.h"
/*!
* \brief Simple demonstration of WarpReduce.
*/

#include <stdio.h>
#include <typeinfo>

#include "cuda_util.h"

#include <cub/warp/warp_reduce.cuh>
#include <cub/util_allocator.cuh>

hipcub::CachingDeviceAllocator  g_allocator(true);

//Initialize problem.
template <typename T>
void Initialize(T *h_in, int num_items) {
  for (int i = 0; i < num_items; ++i)
    h_in[i] = (T) i;
}

// Compute solution.
template <typename T>
void SolveInCPU(T *h_in, T &h_out, int num_items) {
  h_out = (T) 0;
  for (int i = 0; i < num_items; ++i) {
    h_out += h_in[i];
  }
}

enum WarpReduceMode {
  FULL,
  PARTIAL
};

/*
*   The difference between Full and Partial is the input parameters
* of hipcub::WarpReduce::Reduce.
*
*   In the mode Full, you can call the Reduce function like this :
* T output = WarpReduce(temp_storage[warp_id]).Reduce(input, reduction_op);
* And in the mode PARTIAL, then :
* T output = WarpReduce(temp_storage[warp_id]).Reduce(input, reduction_op, valid_warp_threads);
*
*   PS: The Reduce functions in FULL and PARTIAL are implemented differently.
*/
// Full-tile warp reduction.
template <typename T, typename ReductionOp, typename WarpReduce>
static __device__ __forceinline__
  T DeviceReduce(typename WarpReduce::TempStorage &temp_storage,
                 T &data, 
                 ReductionOp &reduction_op,
                 hipcub::Int2Type<FULL> test_mode,
                 const int &valid_warp_threads) {
  return WarpReduce(temp_storage).Reduce(data, reduction_op);
}

// Partially-full warp reduction.
template <typename T, typename ReductionOp, typename WarpReduce>
  static __device__ __forceinline__
  T DeviceReduce(typename WarpReduce::TempStorage &temp_storage,
                 T &data, 
                 ReductionOp &reduction_op,
                 hipcub::Int2Type<PARTIAL> test_mode,
                 const int &valid_warp_threads) {
  return WarpReduce(temp_storage).Reduce(data, reduction_op, valid_warp_threads);
}

/**
 * Warp reduction kernel
 */
template <int NUM_WARPS, int LOGICAL_WARP_THREADS, WarpReduceMode TEST_MODE, 
          typename T, typename ReductionOp>
__global__ void WarpReduceKernel(T *d_in, T *d_out, ReductionOp reduction_op,
                                     clock_t *d_elapsed, int valid_warp_threads) {
  // Cooperative warp-reduce utility type (1 warp)
  typedef hipcub::WarpReduce<T, LOGICAL_WARP_THREADS> WarpReduce;

  // Allocate temp storage in shared memory
  __shared__ typename WarpReduce::TempStorage temp_storage[NUM_WARPS];

  // Per-thread tile data
  T input = d_in[threadIdx.x];

  // Record elapsed clocks
  __threadfence_block();      // workaround to prevent clock hoisting
  clock_t start = clock();
  __threadfence_block();      // workaround to prevent clock hoisting

  // Test warp reduce
  int warp_id = threadIdx.x / LOGICAL_WARP_THREADS;

  /// The key function. 
  /// You can call them directly like the two lines below:
  //T output = WarpReduce(temp_storage[warp_id]).Reduce(input, reduction_op); //Full
  //T output = WarpReduce(temp_storage[warp_id]).Reduce(input, reduction_op, valid_warp_threads); //Partial
  /// Or you can use a template to switch them, like this way:
  T output = DeviceReduce<T, ReductionOp, WarpReduce>(
    temp_storage[warp_id], input, reduction_op, 
    hipcub::Int2Type<TEST_MODE>(), valid_warp_threads);

  // Record elapsed clocks
  __threadfence_block();      // workaround to prevent clock hoisting
  clock_t stop = clock();
  __threadfence_block();      // workaround to prevent clock hoisting

  *d_elapsed = stop - start;

  // Store aggregate
  d_out[threadIdx.x] = (threadIdx.x % LOGICAL_WARP_THREADS == 0) ?
    output :
    input;
}

/**
 * Test warp reduction
 */
template <int NUM_WARPS, int LOGICAL_WARP_THREADS,
          typename T, typename ReductionOp>
void TestReduce(ReductionOp reduction_op,
                int valid_warp_threads = LOGICAL_WARP_THREADS) {

  const int BLOCK_THREADS = LOGICAL_WARP_THREADS * NUM_WARPS;

  printf("%d warps, %d warp threads, %d valid lanes, %s (%d bytes) elements:\n",
    NUM_WARPS,
    LOGICAL_WARP_THREADS,
    valid_warp_threads,
    typeid(T).name(),
    (int) sizeof(T));

  // Allocate host arrays
  T *h_in = new T[BLOCK_THREADS];
  T h_out = 0;
  // Initialize problem.
  Initialize<T>(h_in, BLOCK_THREADS);
  SolveInCPU(h_in, h_out, BLOCK_THREADS);

  std::cout << "input array: ";
  for (int i = 0; i < BLOCK_THREADS; i++) {
    std::cout << h_in[i] << ",";
  }
  std::cout << std::endl;

  // Initialize/clear device arrays
  T *d_in = NULL;
  T *d_out = NULL;
  clock_t *d_elapsed = NULL;

  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * BLOCK_THREADS));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * BLOCK_THREADS));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(clock_t)));
  CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * BLOCK_THREADS, hipMemcpyHostToDevice));
  CubDebugExit(hipMemset(d_out, 0, sizeof(T) * BLOCK_THREADS));

  // Run kernel
  if (valid_warp_threads == LOGICAL_WARP_THREADS) {
    // Run full-warp kernel
    WarpReduceKernel<NUM_WARPS, LOGICAL_WARP_THREADS, FULL> << <1, BLOCK_THREADS >> > (
      d_in,
      d_out,
      reduction_op,
      d_elapsed,
      valid_warp_threads);
  }
  else {
    // Run partial-warp kernel
    WarpReduceKernel<NUM_WARPS, LOGICAL_WARP_THREADS, PARTIAL> << <1, BLOCK_THREADS >> > (
      d_in,
      d_out,
      reduction_op,
      d_elapsed,
      valid_warp_threads);
  }

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
  
  clock_t h_elapsed = NULL;
  hipMemcpy(&h_elapsed, d_elapsed, sizeof(clock_t), hipMemcpyDeviceToHost);
  // Copy the output data from device to cpu.
  T *h_out4cub_temp_arr = new T[BLOCK_THREADS];
  hipMemcpy(h_out4cub_temp_arr, d_out, sizeof(T) * BLOCK_THREADS, hipMemcpyDeviceToHost);

  std::cout << "output array: ";
  for (int i = 0; i < BLOCK_THREADS; i++) {
    std::cout << h_out4cub_temp_arr[i] << ",";
  }
  std::cout << std::endl;

  // Merge the result of each warp.
  T h_out4cub = 0;
  for (int i = 0; i < NUM_WARPS; i++) {
    h_out4cub += h_out4cub_temp_arr[i*LOGICAL_WARP_THREADS + 0];
  }
  std::cout << "Result: (h_out vs h_out4cub) == (" << h_out << " vs " \
    << h_out4cub << "), Elapsed clocks:" << h_elapsed << std::endl << std::endl;

  // Cleanup
  if (h_in) delete[] h_in;
  if (h_out4cub_temp_arr) delete[] h_out4cub_temp_arr;
  if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
  if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
  if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}

int main(int argc, char** argv) {
  int ret = cjmcv_cuda_util::InitEnvironment(0);
  if (ret != 0) {
    printf("Failed to initialize the environment for cuda.");
    return -1;
  }

  // normal:
  //   (unsigned)char / (unsigned) short / (unsigned) int / (unsigned) long long 
  //   float / double
  //
  // vector:
  //   uchar1 
  //   uchar2 / ushort2 / uint2 / ulonglong2
  //   uchar4 / ushort4 / uint4 / ulonglong4
  // 
  // functor:
  //   Sum / Max / Min / ArgMax / CastOp / ...
  TestReduce<1, 32, int>(hipcub::Sum()); 
  TestReduce<1, 32, int>(hipcub::Max()); // It just takes the largest number in the input array.
  TestReduce<1, 32, int>(hipcub::Sum(), 30); // Partial, Only calculate the first 30 Numbers.
  TestReduce<2, 16, int>(hipcub::Sum());
  TestReduce<1, 32, double>(hipcub::Sum());

  cjmcv_cuda_util::CleanUpEnvironment();
  return 0;
}