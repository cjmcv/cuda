#include "hip/hip_runtime.h"
/*!
* \brief gemm: C = A * B.
*/
#include "operator/gemm.h"

namespace cux {
// CUDA version 1: 72 ms
// It is rewrited from GemmCPUv2. 
// bi,bj can be replaced by blockIdx.x,blockIdx.y
// i,j can be replaced by threadIdx.x,threadIdx.y
// so just bk and k left. Grid and block is related to the dst matrix.
//
// \ C[ty, tx] = A[ty, k] * B[k, tx]
// for bk -> bk_num_per_grid
//     for k -> k_num_per_block
//         C[bi*bs + ty, bj*bs + tx] = A[bi*bs + ty, bk*bs + k] * B[k*bs + k, bj*bs + tx]
__global__ void GemmKernelv1(const int M, const int N, const int K, const float ALPHA,
  const float *A, const int lda,
  const float *B, const int ldb,
  float *C, const int ldc) {

  const int block_size = blockDim.x; // The Block is square.
  const int block_num_K = K / block_size;

  float c_sub_acc = 0;
  for (int bk = 0; bk < K / block_size; bk++) {
    for (int k = 0;k < block_size; k++) {
      c_sub_acc += A[(blockIdx.y * block_size + threadIdx.y) * lda + (bk * block_size + k)] *
        B[(bk * block_size + k) * ldb + (blockIdx.x * block_size + threadIdx.x)];
    }
  }

  C[(blockIdx.y * block_size + threadIdx.y) * ldc + (blockIdx.x * block_size + threadIdx.x)] += c_sub_acc;
}

// CUDA version 2.
// Use shared memory.
template <int BLOCK_SIZE>
__global__ void GemmKernelv2(const int M, const int N, const int K, const float ALPHA,
  const float *A, const int lda,
  const float *B, const int ldb,
  float *C, const int ldc) {

  __shared__ float a_shared[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float b_shared[BLOCK_SIZE][BLOCK_SIZE];

  float c_sub_acc = 0;
  // For blocks in grid.
  for (int bk = 0; bk < K / BLOCK_SIZE; bk++) {
    a_shared[threadIdx.y][threadIdx.x] = A[(blockIdx.y * BLOCK_SIZE + threadIdx.y) * lda + (bk * BLOCK_SIZE + threadIdx.x)];
    b_shared[threadIdx.y][threadIdx.x] = B[(bk * BLOCK_SIZE + threadIdx.y) * ldb + (blockIdx.x * BLOCK_SIZE + threadIdx.x)];
    // Wait for data to complete loading to Shared memory.
    __syncthreads();

    // For elements in a block.
    for (int k = 0;k < BLOCK_SIZE; k++) {
      c_sub_acc += a_shared[threadIdx.y][k] * b_shared[k][threadIdx.x];
    }
	  // To prevent the case from happening:
	  // The next round of data is loaded when the data in share memory is not used up.
    __syncthreads();
  }

  C[(blockIdx.y * BLOCK_SIZE + threadIdx.y) * ldc + (blockIdx.x * BLOCK_SIZE + threadIdx.x)] += c_sub_acc;
}

void GemmKernel(const int kernel_id, 
                const int M, const int N, const int K, const float ALPHA,
                const float *A, const int lda,
                const float *B, const int ldb,
                float *C, const int ldc) {
  switch (kernel_id) {
  case 0:
    break;
  case 1:
    break;
  default:
    CUXLOG_ERR("Kernel id not found.");
  }
}

//////////////////
// cuda version.
void GEMM::RunOnDevice() {
  // Time recorder.
  GpuTimer gpu_timer;

  // Input.
  gpu_timer.Start();
  const float *A = A_->GetGpuData();
  const float *B = B_->GetGpuData();
  float *C = C_->GetGpuData();
  gpu_timer.Stop();
  gpu_time_in_record_ = gpu_timer.MilliSeconds();

  const float ALPHA = params_.alpha_;
  const int M = A_->shape()[CuxShape::HEIGHT];
  const int N = A_->shape()[CuxShape::WIDTH];
  const int K = B_->shape()[CuxShape::WIDTH];
  const int lda = N;
  const int ldb = K;
  const int ldc = K;

  // Layout.
  const int block_size = 32;
  dim3 threads_per_block(block_size, block_size);
  dim3 blocks_per_grid(N / threads_per_block.x, M / threads_per_block.y);

  // Warm up.
  gpu_timer.Start();
  GemmKernelv1<< <blocks_per_grid, threads_per_block >> >
    (M, N, K, 1.0, A, lda, B, ldb, C, ldc);
  gpu_timer.Stop();
  gpu_time_warnup_record_ = gpu_timer.MilliSeconds();

  // Run.
  loops_ = 1;
  hipMemset(C, 0, sizeof(float) * M * N);

  gpu_time_kernel_record_.clear();
  gpu_timer.Start();

  GemmKernelv1 << <blocks_per_grid, threads_per_block >> >
    (M, N, K, 1.0, A, lda, B, ldb, C, ldc);

  gpu_timer.Stop();
  gpu_time_kernel_record_.push_back(gpu_timer.MilliSeconds() / loops_);


  // Output.
  gpu_timer.Start();
  CUXLOG_COUT("result: %f.", *C_->GetCpuData());
  gpu_timer.Stop();
  gpu_time_out_record_ = gpu_timer.MilliSeconds();
}
}