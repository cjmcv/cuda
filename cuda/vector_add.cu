#include "hip/hip_runtime.h"
/*!
 * \brief Vector addition: C = A + B. 
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include ""

// Kernel
__global__ void VectorAddKernel(const float *A, const float *B,
                                float *C, int num) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < num) {
    C[i] = A[i] + B[i];
  }
}

int VectorAdd(const float *h_a, const float *h_b, const int num, float *h_c) {
  size_t size = num * sizeof(float);
  // Allocate the device input vector
  hipError_t cuda_err;
  hipError_t cuda_err_a, cuda_err_b, cuda_err_c;
  float *d_a = NULL;
  float *d_b = NULL;
  float *d_c = NULL;

  cuda_err_a = hipMalloc((void **)&d_a, size);
  cuda_err_b = hipMalloc((void **)&d_b, size);
  cuda_err_c = hipMalloc((void **)&d_c, size);
  if (cuda_err_a != hipSuccess || cuda_err_b != hipSuccess || cuda_err_c != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector (error code %s)!\n");
    return 1;
  }

  // Copy the host input vectors in host memory to the device input vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  cuda_err_a = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  cuda_err_b = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
  if (cuda_err_a != hipSuccess || cuda_err_a != hipSuccess) {
    fprintf(stderr, "Failed to copy vector from host to device (error code <%s>, <%s>)!\n", 
      hipGetErrorString(cuda_err_a), hipGetErrorString(cuda_err_b));
    return 1;
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  VectorAddKernel << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, num);
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(cuda_err));
    return 1;
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  cuda_err_c = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  if (cuda_err_c != hipSuccess) {
    fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(cuda_err_c));
    return 1;
  }

  // Verify that the result vector is correct
  for (int i = 0; i < num; ++i) {
    if (fabs(h_a[i] + h_b[i] - h_c[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      return 1;
    }
  }

  // Free device global memory
  cuda_err_a = hipFree(d_a);
  cuda_err_b = hipFree(d_b);
  cuda_err_c = hipFree(d_c);
  if (cuda_err_a != hipSuccess || cuda_err_b != hipSuccess || cuda_err_c != hipSuccess) {
    fprintf(stderr, "Failed to free device vector (error code <%s>, <%s>, <%s>)!\n", 
      hipGetErrorString(cuda_err_a), 
      hipGetErrorString(cuda_err_b),
      hipGetErrorString(cuda_err_c));
    return 1;
  }

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  cuda_err = hipDeviceReset();
  if (cuda_err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(cuda_err));
    return 1;
  }

  printf("Done\n");
  return 0;
}

int main(void) {
  int num = 50000;
  // Print the vector length to be used, and compute its size
  size_t size = num * sizeof(float);

  // Allocate the host input vector
  float *h_a = (float *)malloc(size);
  float *h_b = (float *)malloc(size);
  float *h_c = (float *)malloc(size);
  if (h_a == NULL || h_b == NULL || h_c == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    return 1;
  }

  // Initialize
  for (int i = 0; i < num; ++i) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
  }

  VectorAdd(h_a, h_b, num, h_c);

  // Free host memory
  free(h_a);
  free(h_b);
  free(h_c);
}
