#include "hip/hip_runtime.h"
/*!
 * \brief Vector addition: C = A + B. 
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include ""

#define CUDA_CHECK(condition) \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA_CHECK error in line %d of file %s \
              : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
      exit(EXIT_FAILURE); \
    } \
  } while(0);

// Kernel
__global__ void VectorAddKernel(const float *A, const float *B,
                                float *C, int num) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < num) {
    C[i] = A[i] + B[i];
  }
}

int VectorAdd(const float *h_a, const float *h_b, const int num, float *h_c) {
  size_t size = num * sizeof(float);
  // Allocate the device input vector
  float *d_a = NULL;
  float *d_b = NULL;
  float *d_c = NULL;

  CUDA_CHECK(hipMalloc((void **)&d_a, size));
  CUDA_CHECK(hipMalloc((void **)&d_b, size));
  CUDA_CHECK(hipMalloc((void **)&d_c, size));

  // Copy the host input vectors in host memory to the device input vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  CUDA_CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  VectorAddKernel << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, num);
  CUDA_CHECK(hipGetLastError());

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  CUDA_CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

  // Verify that the result vector is correct
  for (int i = 0; i < num; ++i) {
    if (fabs(h_a[i] + h_b[i] - h_c[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      return 1;
    }
  }

  // Free device global memory
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  CUDA_CHECK(hipDeviceReset());

  printf("Done\n");
  return 0;
}

int main(void) {
  int num = 50000;
  // Print the vector length to be used, and compute its size
  size_t size = num * sizeof(float);

  // Allocate the host input vector
  float *h_a = (float *)malloc(size);
  float *h_b = (float *)malloc(size);
  float *h_c = (float *)malloc(size);
  if (h_a == NULL || h_b == NULL || h_c == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    return 1;
  }

  // Initialize
  for (int i = 0; i < num; ++i) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
  }

  VectorAdd(h_a, h_b, num, h_c);

  // Free host memory
  free(h_a);
  free(h_b);
  free(h_c);
}
