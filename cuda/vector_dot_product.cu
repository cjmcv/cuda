#include "hip/hip_runtime.h"
/*!
* \brief Vector dot product: h_result = SUM(A * B).
*/
#include <iostream>
#include <hip/hip_runtime.h>
#include ""
#include "time.h"

#define CUDA_CHECK(condition) \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA_CHECK error in line %d of file %s \
              : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
      exit(EXIT_FAILURE); \
    } \
  } while(0);

// Initialize the input data.
void GenArray(const int len, float *arr) {
  for (int i = 0; i < len; i++) {
    arr[i] = 1;//(float)rand() / RAND_MAX + (float)rand() / (RAND_MAX*RAND_MAX);
  }
}

// Normal version in cpu as a reference
float VectorDotProductCPU(const float *vec_a, const float *vec_b, const int len) {
  float h_result = 0;
  for (int i = 0; i<len; i++) {
    h_result += vec_a[i] * vec_b[i];
  }
  return h_result;
}

// CUDA kernel
// Multiply to shared memory.
// Accumulate from all of the shared memory to fewer blocks.
template <int BLOCK_SIZE>
__global__ void VectorDotProductKernel(const float *vec_a, const float *vec_b, const int len, float &res) {
  // Prevents memory access across the border.
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
    i < len;
    i += blockDim.x * gridDim.x) {
    __shared__ float smem[BLOCK_SIZE];
    smem[threadIdx.x] = vec_a[i] * vec_b[i];
    __syncthreads();

    int count = BLOCK_SIZE / 2;
    while (count >= 1) {
      if(threadIdx.x < count) {
        smem[threadIdx.x] += smem[count + threadIdx.x];
      }
      // Synchronize the threads within the block,
      // then go to next round together.
      __syncthreads();
      count /= 2;       // !
    }
    
    if(threadIdx.x == 0)
      atomicAdd(&res, smem[0]);
  }
}

float VectorDotProductCUDA(const float *vec_a, const float *vec_b, const int len, float &result) {
  // Time recorder.
  float msec_total = 0.0f;
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  const int threads_per_block = 1024;
  const int blocks_per_grid = (len + threads_per_block - 1) / threads_per_block;

  // Warm up.
  VectorDotProductKernel<threads_per_block> << <blocks_per_grid, threads_per_block >> >
    (vec_a, vec_b, len, result);
  
  // Record the start event
  CUDA_CHECK(hipEventRecord(start, NULL));

  for (int i = 0; i < 100; i++) {
    hipMemset(&result, 0, sizeof(float));
    VectorDotProductKernel<threads_per_block> << <blocks_per_grid, threads_per_block >> >
      (vec_a, vec_b, len, result);
  }

  // Record the stop event
  CUDA_CHECK(hipEventRecord(stop, NULL));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&msec_total, start, stop));

  return msec_total;
}

int main() {
  const int data_len = 1024000;
  const int data_mem_size = sizeof(float) * data_len;
  float *h_vector_a = (float *)malloc(data_mem_size);
  float *h_vector_b = (float *)malloc(data_mem_size);
  if (h_vector_a == NULL || h_vector_b == NULL ) {
    printf("Fail to malloc.\n");
    return 1;
  }
  
  // Initialize 
  srand(0);
  GenArray(data_len, h_vector_a);
  GenArray(data_len, h_vector_b);

  // CPU
  time_t t = clock();
  float h_result = 0;
  for (int i = 0; i < 100; i++)
    h_result = VectorDotProductCPU(h_vector_a, h_vector_b, data_len);
  printf("\nIn cpu version 1, msec_total = %lld, h_result = %f\n", clock() - t, h_result);

  // GPU
  // Allocate memory in host. 
  float msec_total;
  float *d_vector_a = NULL, *d_vector_b = NULL;
  float *d_result = NULL;
  CUDA_CHECK(hipMalloc((void **)&d_vector_a, data_mem_size));
  CUDA_CHECK(hipMalloc((void **)&d_vector_b, data_mem_size));
  CUDA_CHECK(hipMalloc((void **)&d_result, sizeof(float)));

  // Copy host memory to device
  CUDA_CHECK(hipMemcpy(d_vector_a, h_vector_a, data_mem_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_vector_b, h_vector_b, data_mem_size, hipMemcpyHostToDevice));

  msec_total = VectorDotProductCUDA(d_vector_a, d_vector_b, data_len, *d_result);
  
  CUDA_CHECK(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));
  printf("\nIn gpu version 1, msec_total = %f, h_result = %f\n", msec_total, h_result);

  free(h_vector_a);
  free(h_vector_b);

  hipFree(d_vector_a);
  hipFree(d_vector_b);
  hipFree(d_result);
  CUDA_CHECK(hipDeviceReset());

  system("pause");
  return 0;
}
