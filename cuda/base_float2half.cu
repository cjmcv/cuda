#include "hip/hip_runtime.h"
/*!
* \brief Record the basic usage of float2half.
*/
#include <iostream>
#include <hip/hip_fp16.h>
#include "cuda_util.h"


// Taken from:
// https://github.com/dmlc/mshadow/blob/master/mshadow/half.h
union Bits {
  float f;
  int32_t si;
  uint32_t ui;
};
static int const shift = 13;
static int const shiftSign = 16;

static int32_t const infN = 0x7F800000;   // flt32 infinity
static int32_t const maxN = 0x477FE000;   // max flt16 normal as a flt32
static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
static int32_t const signN = 0x80000000;  // flt32 sign bit

static int32_t const infC = infN >> shift;
static int32_t const nanN = (infC + 1) << shift;  // minimum flt16 nan as a flt32
static int32_t const maxC = maxN >> shift;
static int32_t const minC = minN >> shift;
static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
static int32_t const norC = 0x00400;  // min flt32 normal down shifted

static int32_t const maxD = infC - maxC - 1;
static int32_t const minD = minC - subC - 1;

// Host version of device function __float2half_rn()
uint16_t float2half(const float& value) {
  Bits v, s;
  v.f = value;
  uint32_t sign = v.si & signN;
  v.si ^= sign;
  sign >>= shiftSign;  // logical shift
  s.si = mulN;
  s.si = s.f * v.f;  // correct subnormals
  v.si ^= (s.si ^ v.si) & -(minN > v.si);
  v.si ^= (infN ^ v.si) & -((infN > v.si) & (v.si > maxN));
  v.si ^= (nanN ^ v.si) & -((nanN > v.si) & (v.si > infN));
  v.ui >>= shift;  // logical shift
  v.si ^= ((v.si - maxD) ^ v.si) & -(v.si > maxC);
  v.si ^= ((v.si - minD) ^ v.si) & -(v.si > subC);
  return v.ui | sign;
}

float half2float(const uint16_t& value) {
  Bits v;
  v.ui = value;
  int32_t sign = v.si & signC;
  v.si ^= sign;
  sign <<= shiftSign;
  v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
  v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
  Bits s;
  s.si = mulC;
  s.f *= v.si;
  int32_t mask = -(norC > v.si);
  v.si <<= shift;
  v.si ^= (s.si ^ v.si) & mask;
  v.si |= sign;
  return v.f;
}

// Annotation:
// \function, unsigned short __float2half_rn(float x);  -->  hip/device_functions.h
//   It was already presented in CUDA before CUDA 7.5.
//
// \function, __half __float2half(const float a);       -->  hip/hip_fp16.h
//   It was introduced in CUDA 7.5 and does the same with __float2half_rn,
//   but return a half.
//
// \function, __half2 __float2half2_rn(const float a);  -->  hip/hip_fp16.h
//   It returns a half2 which stores two half into an unsigned int. 
//
// \Difference between __float2half_rn and __float2half.
// https://stackoverflow.com/questions/35198856/half-precision-difference-between-float2half-vs-float2half-rn
//   "I found that (for sm_20) the old __float2half_rn() has an additional int16 to int32
//   operation and does a 32bit store. On the other hand, __float2half_() does not have 
//   this conversion and does a 16bit store."
//   __float2half_rn():
//    /*0040*/         I2I.U32.U16 R0, R0;
//    /*0050*/         STL[R2], R0;
//   __float2half():
//    /*0048*/         STL.U16 [R2], R0;

__global__ void ConvertTest() {
  const float flt_in = 1.1234;

  half res_h = __float2half(flt_in);  
  printf("Device version, float -> half: %f -> %hu\n", flt_in, res_h);

  half2 res_h2 = __float2half2_rn(flt_in);
  printf("Device version, float -> half2: %f -> (x: %d, y: %d)\n", res_h2.x, res_h2.y);

  float res_f = __half2float(res_h);
  printf("Device version, half -> float: %hu -> %f\n", res_h, res_f);
}

int main() {
  ConvertTest << <1, 1 >> >();
  hipDeviceSynchronize();

  float flt_in = 1.1234;
  unsigned short res_h = float2half(1.1234);
  printf("Host version, float -> half: %f -> %hu\n", flt_in, res_h);
  float res_f = half2float(res_h);
  printf("Host version, half -> float: %hu -> %f\n", res_h, res_f);

  float2 a;
  return 0;
}