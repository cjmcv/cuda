#include "hip/hip_runtime.h"
/*!
* \brief Record the basic usage of hipOccupancyMaxPotentialBlockSize.
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include ""

#define CUDA_CHECK(condition) \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA_CHECK error in line %d of file %s \
              : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
      exit(EXIT_FAILURE); \
    } \
  } while(0);

// A simple kernel that can be called with any execution configuration.
__global__ void square(int *arr, int len) {
  extern __shared__ int smem[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < len) {
    arr[idx] *= arr[idx];
  }
}

// Run with automatically configured launch.
// It suggests a block size that achieves the best theoretical occupancy. 
// But the occupancy can not be translated directly to performance.
int Run(const int count, int *d_arr) {
  int block_size;
  int min_grid_size;
  int grid_size;
  size_t dynamic_smem_usage = 0;

  CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size,
    &block_size,
    (void*)square,
    dynamic_smem_usage,
    count));

  printf("Minimum grid size for maximum occupancy: %d \n", min_grid_size);
  printf("Suggested block size: %d \n", block_size);
  printf("Blocksize to dynamic shared memory size: %d \n", dynamic_smem_usage);

  // Round up.
  grid_size = (count + block_size - 1) / block_size;

  // Launch and profile.
  square << <grid_size, block_size, dynamic_smem_usage >> > (d_arr, count);

  return 0;
}

int main() {
  const int count = 100000;
  int size = count * sizeof(int);

  // Initialize.
  int *h_arr;
  h_arr = new int[count];
  for (int i = 0; i < count; i += 1) {
    h_arr[i] = i;
  }

  // To prepare data in device.
  int *d_arr;
  CUDA_CHECK(hipMalloc(&d_arr, size));
  CUDA_CHECK(hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice));

  // The key function.
  Run(count, d_arr);

  // Clear for storing the calculation result.
  for (int i = 0; i < count; i += 1) {
    h_arr[i] = 0;
  }
  // Verify the return data.
  CUDA_CHECK(hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost));
  for (int i = 0; i < count; i += 1) {
    if (h_arr[i] != i * i) {
      printf("index: %d, expected: %d, actual: %d", i, i*i, h_arr[i]);
      return 1;
    }
  }

  // Free.
  delete[] h_arr;  
  CUDA_CHECK(hipFree(d_arr));  
  CUDA_CHECK(hipDeviceReset());
  return 0;
}
