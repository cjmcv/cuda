#include "hip/hip_runtime.h"
/*!
 * \brief A simple task consumer using threads and streams 
 *        with all data in Unified Memory.
 */

#include <iostream>
#include <time.h>
#include <vector>
#include <algorithm>
#include <thread>

#include <hip/hip_runtime.h>
#include ""
#include <hipblas.h>

#define CUDA_CHECK(condition) \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA_CHECK error in line %d of file %s \
              : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
      exit(EXIT_FAILURE); \
    } \
  } while(0);

template <typename T>
struct Task {
  unsigned int size_, id_;
  T *data_;
  T *result_;
  T *vector_;

  Task() : size_(0), id_(0), data_(NULL), result_(NULL), vector_(NULL) {};
  Task(unsigned int s) : size_(s), id_(0), data_(NULL), result_(NULL) {}
  ~Task() {}
    
  // allocate unified memory outside of constructor
  void allocate(const unsigned int size, const unsigned int unique_id) {
    id_ = unique_id;
    size_ = size;
    CUDA_CHECK(hipMallocManaged(&data_, sizeof(T)*size_*size_));
    CUDA_CHECK(hipMallocManaged(&result_, sizeof(T)*size_));
    CUDA_CHECK(hipMallocManaged(&vector_, sizeof(T)*size_));
    CUDA_CHECK(hipDeviceSynchronize());

    // populate data_ with random elements
    for (int i = 0; i < size_*size_; i++) {
      data_[i] = double(rand()) / RAND_MAX;
    }

    for (int i = 0; i < size_; i++) {
      result_[i] = 0.;
      vector_[i] = double(rand()) / RAND_MAX;
    }
  }

  void deallocate() {
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(data_));
    CUDA_CHECK(hipFree(result_));
    CUDA_CHECK(hipFree(vector_));
  }
};

// simple host dgemv: assume data_ is in row-major format and square
template <typename T>
void gemv(int m, int n, T alpha, T *A, T *x, T beta, T *result_) {
  // rows
  for (int i = 0; i < n; i++) {
    result_[i] *= beta;

    for (int j = 0; j < n; j++) {
      result_[i] += A[i*n + j] * x[j];
    }
  }
}

template <typename T>
void Execute(Task<T> &t, hipblasHandle_t *handle, hipStream_t *stream, int tid) {
  if (t.size_ < 100) {
    // perform on host
    printf("Task [%d], thread [%d] executing on host (%d)\n", t.id_, tid, t.size_);

    // attach managed memory to a (dummy) stream to allow host access while the device is running
    CUDA_CHECK(hipStreamAttachMemAsync(stream[0], t.data_, 0, hipMemAttachHost));
    CUDA_CHECK(hipStreamAttachMemAsync(stream[0], t.vector_, 0, hipMemAttachHost));
    CUDA_CHECK(hipStreamAttachMemAsync(stream[0], t.result_, 0, hipMemAttachHost));
    // necessary to ensure Async hipStreamAttachMemAsync calls have finished
    CUDA_CHECK(hipStreamSynchronize(stream[0]));
    // call the host operation
    gemv(t.size_, t.size_, 1.0, t.data_, t.vector_, 0.0, t.result_);
  }
  else {
    // perform on device
    printf("Task [%d], thread [%d] executing on device (%d)\n", t.id_, tid, t.size_);
    double one = 1.0;
    double zero = 0.0;

    // attach managed memory to my stream
    hipblasStatus_t status = hipblasSetStream(handle[tid], stream[tid]);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("hipblasSetStream failed. \n ");
    }
    CUDA_CHECK(hipStreamAttachMemAsync(stream[tid], t.data_, 0, hipMemAttachSingle));
    CUDA_CHECK(hipStreamAttachMemAsync(stream[tid], t.vector_, 0, hipMemAttachSingle));
    CUDA_CHECK(hipStreamAttachMemAsync(stream[tid], t.result_, 0, hipMemAttachSingle));
    // call the device operation
    status = hipblasDgemv(handle[tid], HIPBLAS_OP_N, t.size_, t.size_, &one, t.data_, t.size_, t.vector_, 1, &zero, t.result_, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("hipblasSetStream failed. \n ");
    }
  }
}

template <typename T>
void TaskAssignment(std::vector< Task<T> > &task_list, hipblasHandle_t *handle, hipStream_t *stream, int tid, int num_per_thread) {
  for (int i = tid*num_per_thread; i < (tid + 1)*num_per_thread && i < task_list.size(); i++) {
    printf("process: %d, ", i);
    Execute(task_list[i], handle, stream, tid);
  }
}

// populate a list of tasks with random sizes
template <typename T>
void InitialiseTasks(std::vector< Task<T> > &task_list) {
  for (unsigned int i = 0; i < task_list.size(); i++) {
    // generate random size_
    int size_ = std::max((int)((double(rand()) / RAND_MAX)*1000.0), 64);
    task_list[i].allocate(size_, i);
  }
}

template <typename T>
void ReleaseTasks(std::vector< Task<T> > &task_list) {
  printf("release task:\n");
  for (unsigned int i = 0; i < task_list.size(); i++) {
    task_list[i].deallocate();
    printf("%d, ", i);
  }
}

int InitEnvironment(const int dev_id) {
  CUDA_CHECK(hipSetDevice(dev_id));
  hipDeviceProp_t device_prop;
  hipError_t error = hipGetDeviceProperties(&device_prop, dev_id);
  if (device_prop.computeMode == hipComputeModeProhibited) {
    fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    return 1;
  }
  if (error != hipSuccess) {
    printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
  }
  else {
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", dev_id, device_prop.name, device_prop.major, device_prop.minor);
  }
  return 0;
}

int main() {
  InitEnvironment(0);
  // randomise task sizes
  srand(time(NULL));

  // set number of threads
  const int nthreads = 4;

  // number of streams = number of threads.
  hipStream_t *streams = new hipStream_t[nthreads];
  hipblasHandle_t *handles = new hipblasHandle_t[nthreads];

  for (int i = 0; i < nthreads; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
    hipblasStatus_t status = hipblasCreate(&handles[i]);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("Failed to create cublas handle. \n ");
    }
  }

  // Create list of N tasks
  unsigned int N = 40;
  std::vector<Task<double> > task_list(N);
  InitialiseTasks(task_list);

  printf("Executing tasks on host / device\n");
  std::thread *p = new std::thread[nthreads];
  int num_per_thread = (task_list.size() + nthreads - 1) / nthreads;
  for (int tid = 0; tid < nthreads; tid++) {
    p[tid] = std::thread(TaskAssignment<double>, task_list, handles, streams, tid, num_per_thread);
  }
  for (int tid = 0; tid < nthreads; tid++) {
    p[tid].join();
  }
  printf("\nFinish join() \n");

  hipDeviceSynchronize();

  // Destroy CUDA Streams, cuBlas handles
  for (int i = 0; i < nthreads; i++) {
    hipStreamDestroy(streams[i]);
    hipblasDestroy(handles[i]);
  }

  printf("task_list.size() = %d \n", task_list.size());
  // Release tasks.
  ReleaseTasks(task_list);
  task_list.swap(std::vector<Task<double> >());

  return 0;
}